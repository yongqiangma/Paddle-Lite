#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <vector>
#include "lite/backends/cuda/math/pooling.h"
//#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace lite {
namespace cuda {
namespace math {

template <typename PoolProcess, typename T>
__global__ void KernelPool2D(const int nthreads, const T* input_data,
                             const int channels, const int input_height,
                             const int input_width, const int output_height,
                             const int output_width, const int ksize_height,
                             const int ksize_width, const int stride_height,
                             const int stride_width, const int padding_height,
                             const int padding_width, PoolProcess pool_process,
                             bool exclusive, bool adaptive, T* output_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < nthreads;
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart, hend;
    int wstart, wend;
    if (adaptive) {
      hstart = AdaptStartIndex(ph, input_height, output_height);
      hend = AdaptEndIndex(ph, input_height, output_height);

      wstart = AdaptStartIndex(pw, input_width, output_width);
      wend = AdaptEndIndex(pw, input_width, output_width);
    } else {
      hstart = ph * stride_height - padding_height;
      hend = min(hstart + ksize_height, input_height);
      hstart = max(hstart, 0);

      wstart = pw * stride_width - padding_width;
      wend = min(wstart + ksize_width, input_width);
      wstart = max(wstart, 0);
    }

    input_data += (batch_idx * channels + c) * input_height * input_width;
    T ele = pool_process.initial();
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        pool_process.compute(input_data[h * input_width + w], &ele);
      }
    }
    int pool_size = (exclusive || adaptive) ? (hend - hstart) * (wend - wstart)
                                            : ksize_height * ksize_width;
    pool_process.finalize(static_cast<T>(pool_size), &ele);
    output_data[index] = ele;
  }
}


/*
 * All tensors are in NCHW format.
 * Ksize, strides, paddings are two elements. These two elements represent
 * height and width, respectively.
 */
template <typename PoolProcess, typename T>
class Pool2dFunctor<lite::TargetType::kCUDA, PoolProcess, T> {
 public:
  //void operator()(const lite::HIPContext& context,
  void operator()(){

/*
  void operator()(
                  lite::Tensor& input, const std::vector<int>& ksize,
                  const std::vector<int>& strides,
                  const std::vector<int>& paddings, PoolProcess pool_process,
                  bool exclusive, bool adaptive, lite::Tensor* output) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output->dims()[1];
    const int output_height = output->dims()[2];
    const int output_width = output->dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output->mutable_data<T>(TARGET(kCUDA));

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2D<PoolProcess, T><<<grid, threads, 0,  context.exec_stream()>>>(
        nthreads, input_data, input_channels, input_height, input_width,
        output_height, output_width, ksize_height, ksize_width, stride_height,
        stride_width, padding_height, padding_width, pool_process, exclusive,
        adaptive, output_data);
  */
  }
};



/*

template <typename PoolProcess, typename T>
void Pool2dDirectCUDAFunctor<PoolProcess, T>::operator()(
    const T* input, const std::vector<int>& input_shape,
    const std::vector<int>& output_shape, const std::vector<int>& ksize,
    const std::vector<int>& strides, const std::vector<int>& paddings,
    PoolProcess pool_compute, bool exclusive, T* output, hipStream_t stream) {
  const int batch_size = input_shape[0];
  const int input_channels = input_shape[1];
  const int input_height = input_shape[2];
  const int input_width = input_shape[3];
  const int output_channels = output_shape[1];
  const int output_height = output_shape[2];
  const int output_width = output_shape[3];
  const int ksize_height = ksize[0];
  const int ksize_width = ksize[1];
  const int stride_height = strides[0];
  const int stride_width = strides[1];
  const int padding_height = paddings[0];
  const int padding_width = paddings[1];

  int nthreads = batch_size * output_channels * output_height * output_width;
  int blocks = (nthreads + 1024 - 1) / 1024;
  dim3 threads(1024, 1);
  dim3 grid(blocks, 1);

  KernelPool2D<PoolProcess, T><<<grid, threads, 0, stream>>>(
      nthreads, input, input_channels, input_height, input_width, output_height,
      output_width, ksize_height, ksize_width, stride_height, stride_width,
      padding_height, padding_width, pool_compute, exclusive, false, output);
}
*/
template class Pool2dFunctor<lite::TargetType::kCUDA,
                             lite::cuda::math::MaxPool<float>, float>;
template class Pool2dFunctor<lite::TargetType::kCUDA,
                             lite::cuda::math::AvgPool<float>, float>;

}  // namespace math
}  // namespace cuda
}  // namespace lite
}  // namespace paddle

